/*
An example for a perceptron with feed-forward operation capacities
*/

#include "mltrix.cu"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include "../cudatrix/cudatrix.cu"

class FeedForwardPerceptron
{
public:
    double weight;
    double bias;
};

double forwardPass(double dataX, double weight, double bias)
{
    cudatrix::scalarMult(&dataX, &weight); // multiply weight by the input data
    cudatrix::scalarSum(&dataX, &bias);    // sum preactivated data and bias
    mltrix::sigmoid(&dataX);  // activate our data!
    return dataX;
}

int main()
{
    FeedForwardPerceptron myPerceptron;
    myPerceptron.bias = ((double)rand() / (RAND_MAX)); // random numbers between 1 and 0
    myPerceptron.weight = ((double)rand() / (RAND_MAX));
    double dataToPass = ((double)rand() / (RAND_MAX)); // random X data
    printf("Bias: %f\n", myPerceptron.bias);
    printf("Weight: %f\n", myPerceptron.weight);
    printf("Passed data: %f\n", dataToPass);
    double perceptronOutput = forwardPass(dataToPass, myPerceptron.weight, myPerceptron.bias); // forward pass!
    printf("Perceptron output: %f\n", perceptronOutput);
}