#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <hip/hip_runtime.h>
#define E 2.71828182845904523536
#include <math.h>
#include "stdio.h"

__device__ double sigmoidOp(double a)
{
    return pow(E, a) / (pow(E, a) + 1);
}

__global__ void kernelSigmoid(double *numberToTransform)
{
    *numberToTransform = sigmoidOp(*numberToTransform);
}

void mltrix::sigmoid(double *numberToTransform)
{
    double *a_d;
    hipMallocManaged(&a_d, 1 * sizeof(double));
    hipMemcpy(a_d, numberToTransform, 1 * sizeof(double), hipMemcpyHostToDevice);
    kernelSigmoid<<<1, 1>>>(a_d);
    hipDeviceSynchronize();
    *numberToTransform = *a_d;
    hipFree(a_d);
}
